#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include "00_rtc.cu"
#include <cuda_pipeline.h>
#include <cuda/barrier>
#include <cuda/ptx>




// a: row major
// b: col major
// c: row major
// template <int elts, bool print>
constexpr int elts = 16 * 16;
constexpr bool print = true;
__global__ void mma_ptx_kernel(half *c_ptr, half *a_ptr, half *b_ptr, half *d_ptr) {
    int tid = threadIdx.x;

    // ldgsts
    __shared__ half smem_a[elts];
    __shared__ half smem_b[elts];

    half *src = a_ptr + tid / 2 * 16 + tid % 2 * 8;
    half *dst = smem_a + tid * 8;
    uint32_t addr = __cvta_generic_to_shared(dst);
    asm("cp.async.cg.shared.global [%0], [%1], 16;\n"
        :
        :"r"(addr), "l"(src)
    );
    asm("cp.async.commit_group;\n"::);

    src = b_ptr + tid / 2 * 16 + tid % 2 * 8;
    dst = smem_b + tid * 8;
    addr = __cvta_generic_to_shared(dst);
    asm("cp.async.cg.shared.global [%0], [%1], 16;\n"
        :
        :"r"(addr), "l"(src)
    );
    asm("cp.async.commit_group;\n"::);

    asm("cp.async.wait_group 0;\n"::);
    __syncthreads();

    if(tid == 0 && print) {
        printf("\nsmem_a:\n");
        print_mem(smem_a);


        printf("\nsmem_b:\n");
        print_mem(smem_b);

    }

    // ldmatrix
    uint32_t a_regs[4];
    uint32_t b_regs[4];
    src = smem_a + tid % 16 * 16 + tid / 16 * 8;
    /*	mul.wide.s32 	%rd57, %r97, 2;
        add.s64 	%rd59, %rd27, %rd57;
        cvt.u32.u64 	%r83, %rd59;*/

    addr = __cvta_generic_to_shared(src);
    asm("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0, %1, %2, %3}, [%4];\n"
        :"=r"(a_regs[0]), "=r"(a_regs[1]), "=r"(a_regs[2]), "=r"(a_regs[3])
        :"r"(addr)
    );

    src = smem_b + tid % 16 * 16 + tid / 16 * 8;
    addr = __cvta_generic_to_shared(src);
    asm("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0, %1, %2, %3}, [%4];\n"
        :"=r"(b_regs[0]), "=r"(b_regs[1]), "=r"(b_regs[2]), "=r"(b_regs[3])
        :"r"(addr)
    );

    if(tid == 0 && print) {
        printf("\na_regs:\n");

        for(int i = 0; i < 4; i ++) {
            half *data = reinterpret_cast<half*>(&a_regs[i]);
            printf("%6.2lf, %6.2lf\n", __half2float(data[0]), __half2float(data[1]));
        }

        printf("\nb_regs:\n");
        for(int i = 0; i < 4; i ++) {
            half *data = reinterpret_cast<half*>(&b_regs[i]);
            printf("%6.2lf, %6.2lf\n", __half2float(data[0]), __half2float(data[1]));
        }
    }

    // mma
    float accum[8] = {0};

    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
        "{%0, %1, %2, %3}, "
        "{%4, %5, %6, %7}, "
        "{%8, %9}, "
        "{%10, %11, %12, %13};\n"
        : "=f"(accum[0]),"=f"(accum[1]),"=f"(accum[4]),"=f"(accum[5])
        : "r"(a_regs[0]),"r"(a_regs[1]),"r"(a_regs[2]),"r"(a_regs[3]),
          "r"(b_regs[0]),"r"(b_regs[2]),
          "f"(accum[0]),"f"(accum[1]),"f"(accum[4]),"f"(accum[5])
    );

    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
        "{%0, %1, %2, %3}, "
        "{%4, %5, %6, %7}, "
        "{%8, %9}, "
        "{%10, %11, %12, %13};\n"
        : "=f"(accum[2]),"=f"(accum[3]),"=f"(accum[6]),"=f"(accum[7])
        : "r"(a_regs[0]),"r"(a_regs[1]),"r"(a_regs[2]),"r"(a_regs[3]),
          "r"(b_regs[1]),"r"(b_regs[3]),
          "f"(accum[2]),"f"(accum[3]),"f"(accum[6]),"f"(accum[7])
    );
    
    if(tid == 0 && print) {
        printf("\naccum:\n");
        for(int i = 0; i < 8; i ++) {
            printf("%.2lf ", accum[i]);
        }
        printf("\n");
    }

    // sts
    __shared__ half smem_c[elts];
    int row = tid / 4;
    int col = tid % 4 * 2;
    for(int i = 0; i < 2; i ++) {
        for(int j = 0; j < 2; j ++) {
            smem_c[(i*8+row)*16+(j*8+col)+0] = __float2half(accum[i*4+j*2+0]);
            smem_c[(i*8+row)*16+(j*8+col)+1] = __float2half(accum[i*4+j*2+1]);
        }
    }
    __syncthreads();

    // stg
    src = smem_c + tid / 2 * 16 + tid % 2 * 8;
    dst = c_ptr + tid / 2 * 16 + tid % 2 * 8;
    for(int i = 0; i < 8; i ++) {
        dst[i] = src[i];
    }
    if(tid == 0){
        printf("\nc_ptr:\n");
        print_mem(c_ptr);
        printf("\nd_ptr:\n");
        print_mem(d_ptr);
    }
}


using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;


__launch_bounds__(32*4)
__global__ void tma_1d_kernel(half* ptr, int elts)
{
  // Shared memory buffer. The destination shared memory buffer of
  // a bulk operations should be 16 byte aligned.
 extern __shared__ __align__(16) half smem[];
  
  ////////////////// global mem -> shared mem //////////////////
  // 1. a) Initialize shared memory barrier with the number of threads participating in the barrier.
  //    b) Make initialized barrier visible in async proxy.
  #pragma nv_diag_suppress static_var_with_dynamic_init
  __shared__ barrier bar;
  if (threadIdx.x == 0) { 
/*  mov.u64 	%rd9, _ZZ13tma_1d_kernelP6__halfiE3bar;
	cvt.u32.u64 	%r10, %rd9;
    mbarrier.init.shared.b64 [%r10], %r11;*/
    init(&bar, blockDim.x);                      // a)
/*  fence.proxy.async.shared::cta; */
    cde::fence_proxy_async_shared_cta();         // b)
  }
/* bar.sync 	0; */
  __syncthreads();

  // 2. Initiate TMA transfer to copy global to shared memory.
  if (threadIdx.x == 0) {
    // 3a. cuda::memcpy_async arrives on the barrier and communicates
    //     how many bytes are expected to come in (the transaction count)
/*
 * cp.async.bulk.shared::cluster.global.mbarrier::complete_tx::bytes [%r12], [%rd10], %r16, [%r15]
 * mbarrier.expect_tx.relaxed.cta.shared::cta.b64 [%r15], %r16;
*/
    cuda::memcpy_async(
        smem, 
        ptr,
        cuda::aligned_size_t<16>(sizeof(half)*elts),
        bar
    );
  }
  // 3b. All threads arrive on the barrier
/*
  mbarrier.arrive.shared::cta.b64                             %rd13,  [%r17], %r18; 
*/
  barrier::arrival_token token = bar.arrive();
  
  // 3c. Wait for the data to have arrived.
/* ??? wait + sleep? .............. 这么长的代码?
 * 	mov.u64 %rd14, %globaltimer;
 * 	mbarrier.try_wait.shared.b64 p, [%r17], %rd13;	
 *  selp.b32 %r20, 1, 0, p;
 * 	
*/
  bar.wait(std::move(token));

  // 4. Compute saxpy and write back to shared memory
  for (int i = threadIdx.x; i < elts; i += blockDim.x) {
  /*add.f16 %rs2,%rs3,%rs1*/
    smem[i] = __hadd(smem[i], __float2half(1.0));
  }
  
  ////////////////// shared mem -> global mem //////////////////
  // 5. Wait for shared memory writes to be visible to TMA engine.
// 同上
  cde::fence_proxy_async_shared_cta();   // b)
  __syncthreads();
  // After syncthreads, writes by all threads are visible to TMA engine.

  if(threadIdx.x == 0) {
    printf("\ndata on device: %d\n", elts);
    print_mem(smem, 32, 32);
    // for(int i = 0; i < elts; i ++) {
    //     printf("%.2lf ", __half2float(smem[i]));
    // }
  }

  // 6. Initiate TMA transfer to copy shared memory to global memory
  if (threadIdx.x == 0) {
/*	cp.async.bulk.global.shared::cta.bulk_group [%rd36], [%r62], %r63; */
    cde::cp_async_bulk_shared_to_global(
            ptr, smem, sizeof(half)*elts);
    // 7. Wait for TMA transfer to have finished reading shared memory.
    // Create a "bulk async-group" out of the previous bulk copy operation.
/*  cp.async.bulk.commit_group;*/
    cde::cp_async_bulk_commit_group();
    // Wait for the group to have completed reading from shared memory.
/*  cp.async.bulk.wait_group.read 0 */
    cde::cp_async_bulk_wait_group_read<0>();
  }
}


